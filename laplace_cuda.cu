/*
*******************************************************************
* Laplace Serial Version
*
* T is initerially 0.0
* Boundaries are as follows

*             T = 0              
*     |-------------------| 0    
*     |                   |      
*  T  |                   |      
*  =  |                   |  T   
*  0  |                   |      
*     |                   |      
*     |                   |      
*     |                   |      
*     |-------------------| 100  
*     0                 100
*
* Use Central Differencing Method
*
*Modified by Shiva Gopalakkrishnan, NPS...2010
*Modified by  John Urbanic,  PSC.... 2008;
* From the original Fortran version by Sushell Chitre, R Reddy, PSC 1994
*
******************************************************************   */

#define NC       510                   /* Number of Cols        */
#define NR       510                   /* Number of Rows        */
#define NITER    1000                   /* Max num of Iterations */
#define MAX(x,y) ( ((x) > (y)) ? x : y )

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h> /* only for timing */
#include "hip/hip_runtime.h"

void initialize( float t[NR+2][NC+2] );
void set_bcs   ( float t[NR+2][NC+2]);
void print_trace( float t[NR+2][NC+2], int iter);

__global__ void compute( float *t, float *told);
__global__ void update( float *t, float *told);

int main( int argc, char **argv ){
  
  int        niter;                     /* iter counter  */

  float      t[NR+2][NC+2];            /*temperature */
  float      told[NR+2][NC+2];         /* previous temperature*/
  float      dt;                        /* Delta t       */
  clock_t    start, end;              /* timiing */
  int        i, j, iter;
  double  elapsed;

  float *dev_t, *dev_told; //device pointers
  int size = (NR+2)*(NC+2)*sizeof(float); 

    start = clock();
  hipMalloc((void **)&dev_t,size);
  hipMalloc((void **)&dev_told,size);


    initialize(t);                  /* Set initial guess to 0 */

  set_bcs(t);         /* Set the Boundary Conditions */

  for( i=0; i<=NR+1; i++ )       /* Copy the values into told */
    for( j=0; j<=NC+1; j++ )
      told[i][j] = t[i][j];

  //   printf("How many iterations [100-1000]?\n");
  // scanf("%d", &niter);
  niter=1000;
   if( niter>NITER ) niter = NITER;
 
 
/*----------------------------------------------------------*
 |       Do Computation on grid for Niter iterations    |
 *----------------------------------------------------------*/
 hipMemcpy(dev_told,&told,size,hipMemcpyHostToDevice);

  for( iter=1; iter<=niter; iter++ ) {
    
   

    compute<<<512,512 >>>(dev_t,dev_told);

    update<<<512,512 >>>(dev_t,dev_told);



/*   Periodically print some test Values   */
    if( (iter%100) == 0 ) {
    hipMemcpy(&t,dev_t,size,hipMemcpyDeviceToHost);
      print_trace( t, iter );
    }

  }  /* End of kernal */
  end = clock();
  elapsed = ((double) (end - start)) / CLOCKS_PER_SEC;   
   printf(" \n Time taken is %f \n",elapsed);

}    /* End of Program   */


__global__ void compute(float *t, float *told)

{
  //Write CUDA kernal here
  int thread = threadIdx.x;
  int block = blockIdx.x;
  int dimen = blockDim.x;

  int i = thread + block*dimen;
  int j1 = thread + (block -1)*dimen;
  int j2 = thread + (block+1)*dimen;

  if((block != 0) && (block != 511))
    {
      if((thread !=0) && (thread !=511))
	{
	  t[i]=0.25*(told[i-1]+told[i+1]+told[j1]+told[j2]);
	}
      else
	{
      t[i]=told[i];
	}
    }
  else
    {
      t[i]=told[i];
    }

}

__global__ void update(float *t, float *told)

{
  //Write CUDA kernal here
  int thread = threadIdx.x;
  int block = blockIdx.x;
  int dimen = blockDim.x;

  int i = thread + block*dimen;
  told[i]=t[i];
  

}

/*******************************************************************
 *								    *
 * Initialize all the values to 0. as a starting value              *
 *								    *
 ********************************************************************/

void initialize( float t[NR+2][NC+2] ){

  int        i, j, iter;
	  
  for( i=0; i<=NR+1; i++ )
    for ( j=0; j<=NC+1; j++ )
      t[i][j] = 0.0;
}



/********************************************************************
 *								    *
 * Set the values at the boundary.  Values at the boundary do not   *
 * change throughout the execution of the program		    *
 *								    *
 ********************************************************************/

void set_bcs( float t[NR+2][NC+2]){

  int i, j;

  for( i=0; i<=NR+1; i++){
    t[i][0] = 0.0;
    t[i][NC+1] = (100.0/NR)*i;
  }

  for( j=0; j<=NC+1; j++){
    t[0][j] = 0.0;
    t[NR+1][j] = (100.0/NR)*j;
  }

}



/********************************************************************
 *								    *
 * Print the trace only in the bottom corner where most action is   *
 *								    *
 ********************************************************************/

void print_trace( float t[NR+2][NC+2], int iter ){

  int joff, i;
  joff=0;
    printf("\n---------- Iteration number: %d ------------\n", iter);

    for(i=NR-10; i<=NR; i++){
      printf("%15.8f", t[i][joff+i]);
    }

  return;
}
