//HW3
//130010054
#include <stdio.h>
#include <math.h>
#include <time.h>
#include "hip/hip_runtime.h"
#define TILE_WIDTH 4
__global__ void
MatrixMulSh( float *Md , float *Nd , float *Pd , const int WIDTH )
{
  __shared__ float Mds [TILE_WIDTH][TILE_WIDTH] ;
  __shared__ float Nds [TILE_WIDTH][TILE_WIDTH] ;
  unsigned int col = TILE_WIDTH*blockIdx.x + threadIdx.x ;
  unsigned int row = TILE_WIDTH*blockIdx.y + threadIdx.y ;
  for (int m = 0 ; m<WIDTH/TILE_WIDTH ; m++ )
  {
    Mds[threadIdx.y][threadIdx.x] =  Md[row*WIDTH + (m*TILE_WIDTH + threadIdx.x)]  ;
    Nds[threadIdx.y][threadIdx.x] =  Nd[ ( m*TILE_WIDTH + threadIdx.y) * WIDTH + col] ;
    __syncthreads();
    for ( int k = 0; k<TILE_WIDTH ; k++ )
      Pd[row*WIDTH + col]+= Mds[threadIdx.x][k] * Nds[k][threadIdx.y] ;
    __syncthreads();
  }
}
int main ()
{
  const int WIDTH = 10000;
  clock_t  start,end;
  double elapsed;
  float A[WIDTH][WIDTH] ,B[WIDTH][WIDTH],C1[WIDTH][WIDTH] ,C[WIDTH][WIDTH]  ;
  float *A_d , *B_d ,*C1_d,*C_d ; // device array
  int i , j ;
  for ( i = 0 ; i<WIDTH ; i++ )
  {
    for (j = 0 ; j<WIDTH ; j++ )
    {
      A[i][j] = i+j+2 ;
      B[i][j] = (i+1)*(j+1);
    }
  }
  start = clock();
  hipMalloc((void **) &A_d , WIDTH*WIDTH*sizeof (int) ) ;
  hipMalloc((void **) &B_d , WIDTH*WIDTH*sizeof (int) ) ;
  hipMemcpy ( A_d , A , WIDTH*WIDTH*sizeof (int) , hipMemcpyHostToDevice ) ;
  hipMemcpy ( B_d , B , WIDTH*WIDTH*sizeof (int) , hipMemcpyHostToDevice ) ;
  hipMalloc((void **) &C1_d , WIDTH*WIDTH*sizeof (int) ) ;
  hipMalloc((void **) &C_d , WIDTH*WIDTH*sizeof (int) ) ;
  dim3 dimGrid ( WIDTH/TILE_WIDTH , WIDTH/TILE_WIDTH ,1 ) ;
  dim3 dimBlock( TILE_WIDTH, TILE_WIDTH, 1 ) ;
  MatrixMulSh<<<dimGrid,dimBlock>>> ( A_d , B_d ,C_d , WIDTH) ;
  hipMemcpy(C , C_d , WIDTH*WIDTH*sizeof(int) ,hipMemcpyDeviceToHost) ;
  end = clock();
  elapsed = ((double)(end - start)) / CLOCKS_PER_SEC;
  printf("\n Time taken is %f\n",elapsed);
  system("pause") ;
}
